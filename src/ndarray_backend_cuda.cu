#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <pybind11/numpy.h>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>

#include <iostream>
#include <sstream>

namespace needle {
namespace cuda {

#define BASE_THREAD_NUM 256
#define TILE 4

const size_t L = 8 * TILE; // => 8 * 8 = 64 threads; 16 * 16 = 256 threads
const size_t S = 8 * TILE; // => 16 * 4 * 8 * 4 * 4-byte = 8k shared memory

typedef float scalar_t;
const size_t ELEM_SIZE = sizeof(scalar_t);

struct CudaArray {
  CudaArray(const size_t size) {
    hipError_t err = hipMalloc(&ptr, size * ELEM_SIZE);
    if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err));
    this->size = size;
  }
  ~CudaArray() { hipFree(ptr); }
  size_t ptr_as_int() { return (size_t)ptr; }
  
  scalar_t* ptr;
  size_t size;
};

struct CudaDims {
  dim3 block, grid;
};

CudaDims CudaOneDim(size_t size) {
  /**
   * Utility function to get cuda dimensions for 1D call
   */
  CudaDims dim;
  size_t num_blocks = (size + BASE_THREAD_NUM - 1) / BASE_THREAD_NUM;
  dim.block = dim3(BASE_THREAD_NUM, 1, 1);
  dim.grid = dim3(num_blocks, 1, 1);
  return dim;
}

#define MAX_VEC_SIZE 8
struct CudaVec {
  uint32_t size;
  uint32_t data[MAX_VEC_SIZE];
};

CudaVec VecToCuda(const std::vector<uint32_t>& x) {
  CudaVec shape;
  if (x.size() > MAX_VEC_SIZE) {
    throw std::runtime_error("Exceeded CUDA supported max dimesions");
  }
  shape.size = x.size();
  for (size_t i = 0; i < x.size(); i++) {
    shape.data[i] = x[i];
  }
  return shape;
}

////////////////////////////////////////////////////////////////////////////////
// Fill call
////////////////////////////////////////////////////////////////////////////////

__global__ void FillKernel(scalar_t* out, scalar_t val, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = val;
}

void Fill(CudaArray* out, scalar_t val) {
  CudaDims dim = CudaOneDim(out->size);
  FillKernel<<<dim.grid, dim.block>>>(out->ptr, val, out->size);
}

////////////////////////////////////////////////////////////////////////////////
// Compact and setitem cals
////////////////////////////////////////////////////////////////////////////////

// Untility function to convert contiguous index i to memory location from strides

__global__ void CompactKernel(const scalar_t* a, scalar_t* out, size_t size, 
    CudaVec shape, CudaVec strides, size_t offset) {
  /**
   * The CUDA kernel for the compact operation.
   * This should effectively map a single entry in the 
   * non-compact input a, to the corresponding item (at location gid) in the compact array out.
   * 
   * Args:
   *   a: CUDA pointer to a array
   *   out: CUDA point to out array
   *   size: size of out array
   *   shape: vector of shapes of a and out arrays (of type CudaVec, for past passing to CUDA kernel)
   *   strides: vector of strides of out array
   *   offset: offset of out array
   */
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) {
    /// BEGIN YOUR SOLUTION
    size_t a_idx = 0; 
    size_t remain = gid;
    size_t stride = size;
    size_t indexx = 0;
    for(size_t i = 0; i < shape.size; i++) {
      stride = stride / shape.data[i];
      indexx = remain / stride;
      remain = remain % stride;
      a_idx += strides.data[i] * indexx;
    }
    out[gid] = a[a_idx + offset];
    /// END YOUR SOLUTION
  }
}

void Compact(const CudaArray& a, CudaArray* out, std::vector<uint32_t> shape,
             std::vector<uint32_t> strides, size_t offset) {
  /**
   * Compact an array in memory. Unlike the C++ version, in CUDA this will 
   * primarily call the relevant CUDA kernel. In this case, we illustrate 
   * how you should set this up (i.e., we give you the code for this fuction, 
   * and also the prototype for the CompactKernel() function).  For the functions 
   * after this, however, you'll need to define these kernels as you see fit to 
   * execute the underlying function.
   * 
   * Args:
   *   a: non-compact represntation of the array, given as input
   *   out: compact version of the array to be written
   *   shape: shapes of each dimension for a and out
   *   strides: strides of the *a* array (not out, which has compact strides)
   *   offset: offset of the *a* array (not out, which has zero offset)
   */
  CudaDims dim = CudaOneDim(out->size);
  CompactKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size, 
    VecToCuda(shape), VecToCuda(strides), offset);
}


__global__ void EwiseSetitemKernel(const scalar_t* a, scalar_t* out, size_t size, 
    CudaVec shape, CudaVec strides, size_t offset) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) {
    /// BEGIN YOUR SOLUTION
    size_t out_idx = 0;
    size_t remain = gid;
    size_t stride = size;
    size_t indexx = 0;
    // 
    for(size_t i = 0; i < shape.size; i++) {
      stride = stride / shape.data[i];
      indexx = remain / stride;
      remain = remain % stride;
      out_idx += strides.data[i] * indexx;
    }
    out[out_idx + offset] = a[gid];
    /// END YOUR SOLUTION
  }
}

void EwiseSetitem(const CudaArray& a, CudaArray* out, std::vector<uint32_t> shape,
                  std::vector<uint32_t> strides, size_t offset) {
  /**
   * Set items in a (non-compact) array using CUDA. 
   * You will most likely want to implement a EwiseSetitemKernel() function, 
   * similar to those above, that will do the actual work.
   * 
   * Args:
   *   a: _compact_ array whose items will be written to out
   *   out: non-compact array whose items are to be written
   *   shape: shapes of each dimension for a and out
   *   strides: strides of the *out* array (not a, which has compact strides)
   *   offset: offset of the *out* array (not a, which has zero offset)
   */
  /// BEGIN YOUR SOLUTION
  CudaDims dim = CudaOneDim(out->size);
  EwiseSetitemKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, a.size, 
    VecToCuda(shape), VecToCuda(strides), offset);  
  /// END YOUR SOLUTION
}


__global__ void ScalarSetitemKernel(const scalar_t val, scalar_t* out, size_t size, 
    CudaVec shape, CudaVec strides, size_t offset) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) {
    /// BEGIN YOUR SOLUTION
    size_t out_idx = 0;
    size_t remain = gid;
    size_t stride = size;
    size_t indexx = 0;
    // 
    for(size_t i = 0; i < shape.size; i++) {
      stride = stride / shape.data[i];
      indexx = remain / stride;
      remain = remain % stride;
      out_idx += strides.data[i] * indexx;
    }
    out[out_idx + offset] = val;
    /// END YOUR SOLUTION
  }
}


void ScalarSetitem(size_t size, scalar_t val, CudaArray* out, std::vector<uint32_t> shape, std::vector<uint32_t> strides, size_t offset) {
  /**
   * Set items is a (non-compact) array
   * 
   * Args:
   *   size: number of elements to write in out array (note that this will not be 
   *      the same as out.size, because out is a non-compact subset array);
   *      it _will_ be the same as the product of items in shape, 
   *      but covenient to just pass it here.
   *   val: scalar value to write to
   *   out: non-compact array whose items are to be written
   *   shape: shapes of each dimension of out
   *   strides: strides of the out array
   *   offset: offset of the out array
   */
  /// BEGIN YOUR SOLUTION
  CudaDims dim = CudaOneDim(out->size);
  ScalarSetitemKernel<<<dim.grid, dim.block>>>(val, out->ptr, size, 
    VecToCuda(shape), VecToCuda(strides), offset);
  /// END YOUR SOLUTION
}

////////////////////////////////////////////////////////////////////////////////
// Elementwise and scalar operations
////////////////////////////////////////////////////////////////////////////////

__global__ void EwiseAddKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] + b[gid];
}

void EwiseAdd(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  /**
   * Add together two CUDA array
   */
  CudaDims dim = CudaOneDim(out->size);
  EwiseAddKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarAddKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] + val;
}

void ScalarAdd(const CudaArray& a, scalar_t val, CudaArray* out) {
  /**
   * Add together a CUDA array and a scalar value.
   */
  CudaDims dim = CudaOneDim(out->size);
  ScalarAddKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

/**
 * In the code the follows, use the above template to create analogous elementise
 * and and scalar operators for the following functions.  See the numpy backend for
 * examples of how they should work.
 *   - EwiseMul, ScalarMul
 *   - EwiseDiv, ScalarDiv
 *   - ScalarPower

 *   - EwiseMaximum, ScalarMaximum
 *   - EwiseEq, ScalarEq
 *   - EwiseGe, ScalarGe

 *   - EwiseLog
 *   - EwiseExp
 *   - EwiseTanh
 *
 * If you implement all these naively, there will be a lot of repeated code, so
 * you are welcome (but not required), to use macros or templates to define these
 * functions (however you want to do so, as long as the functions match the proper)
 * signatures above.
 */

// https://docs.nvidia.com/cuda/cuda-math-api/group__CUDA__MATH__SINGLE.html#group__CUDA__MATH__SINGLE

/// BEGIN YOUR SOLUTION
__global__ void EwiseExpKernel(const scalar_t* a, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) { out[gid] = expf(a[gid]); }
}

void EwiseExp(const CudaArray& a, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  EwiseExpKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size);
}

__global__ void EwiseTanhKernel(const scalar_t* a, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) { out[gid] = tanhf(a[gid]); }
}

void EwiseTanh(const CudaArray& a, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  EwiseTanhKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size);
}

__global__ void EwiseLogKernel(const scalar_t* a, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) { out[gid] = logf(a[gid]); }
}

void EwiseLog(const CudaArray& a, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  EwiseLogKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size);
}


__global__ void EwiseGeKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) { out[gid] = a[gid] > b[gid] ? 1.0 : 0.0; }
}

void EwiseGe(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  EwiseGeKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarGeKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) { out[gid] = a[gid] >= val ? 1.0 : 0.0; }
}

void ScalarGe(const CudaArray& a, scalar_t val, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  ScalarGeKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}


__global__ void EwiseEqKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) { out[gid] = a[gid] == b[gid] ? 1.0 : 0.0; }
}

void EwiseEq(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  EwiseEqKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarEqKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) { out[gid] = a[gid] == val ? 1.0 : 0.0; }
}

void ScalarEq(const CudaArray& a, scalar_t val, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  ScalarEqKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}


__global__ void EwiseMaximumKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) { out[gid] = fmaxf(a[gid], b[gid]); }
}

void EwiseMaximum(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  EwiseMaximumKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarMaximumKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) { out[gid] = fmaxf(a[gid], val); }
}

void ScalarMaximum(const CudaArray& a, scalar_t val, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  ScalarMaximumKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}


__global__ void ScalarPowerKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) { out[gid] = powf(a[gid], val); }
}

void ScalarPower(const CudaArray& a, scalar_t val, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  ScalarPowerKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

__global__ void EwiseMulKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) { out[gid] = a[gid] * b[gid]; }
}

void EwiseMul(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  EwiseMulKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarMulKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) { out[gid] = a[gid] * val; }
}

void ScalarMul(const CudaArray& a, scalar_t val, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  ScalarMulKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

__global__ void EwiseDivKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) { out[gid] = a[gid] / b[gid]; }
}

void EwiseDiv(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  EwiseDivKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarDivKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) { out[gid] = a[gid] / val; }
}

void ScalarDiv(const CudaArray& a, scalar_t val, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  ScalarDivKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}
/// END YOUR SOLUTION

////////////////////////////////////////////////////////////////////////////////
// Elementwise and scalar operations
////////////////////////////////////////////////////////////////////////////////

__global__ void SimpleMatmulKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, uint32_t M, uint32_t N, uint32_t P) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < M * P) {
    const size_t i = gid / P;
    const size_t j = gid % P;
    scalar_t tmp = 0;
    for (size_t k = 0; k < N; k++) {
      tmp += a[i * N + k] * b[k * P + j];
    }
    out[gid] = tmp;
  }
}


__global__ void MatmulTiledKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size, uint32_t N, uint32_t P) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) {
    /// BEGIN YOUR SOLUTION
    const size_t P_T = P / TILE;
    const size_t ybase = (gid / P_T) * TILE;
    const size_t xbase = (gid % P_T) * TILE;
    
    float c_t[TILE][TILE], a_t[TILE], b_t[TILE];
    for (size_t i = 0; i < TILE; ++i)
      for (size_t j = 0; j < TILE; ++j)
        c_t[i][j] = 0;

    for (size_t k = 0; k < N; ++k) {
      // Khởi tạo mảng a_t, b_t
      for (size_t o = 0; o < TILE; ++o) { 
        a_t[o] = a[(ybase + o)*N + k];
        b_t[o] = b[k*P + (xbase + o)];
      }
      // Tính toán trên local vars
      for (size_t i = 0; i < TILE; ++i)
        for (size_t j = 0; j < TILE; ++j)
          c_t[i][j] += a_t[i] * b_t[j];
    }
    // Update kết quả
    for (size_t i = 0; i < TILE; ++i)
      for (size_t j = 0; j < TILE; ++j)
        out[(ybase + i)*P + (xbase + j)] = c_t[i][j];
    /// END YOUR SOLUTION
  }
}

__global__ void MatmulSharedMemKernel(const scalar_t* a, const scalar_t* b, 
  scalar_t* out, uint32_t P, uint32_t N) {

  // https://youtu.be/jYCxVirq4d0?t=2113
  // out là ma trận C trong video trên gồm M hàng, P cột,
  // xử lý theo block (L,L), L=16*TILE
  // dữ liệu lấy từ A là khối (L,S), từ B là khối (S, L)

  // Mỗi thread nhân sub-matrix(TILE, TILE)
  // Như trong trục tọa độ 2 chiều thì thì x trục tung = hàng, y trục dọc = cột

  // tới vị trí đầu của block
  const size_t yblock = blockIdx.y * blockDim.y * L;
  const size_t xblock = blockIdx.x * blockDim.x * L;
  
  float c_t[TILE][TILE], a_t[TILE], b_t[TILE];
  // local vars will be mapped to registers <= https://youtu.be/jYCxVirq4d0?t=1811
  for (size_t i = 0; i < TILE; ++i)
    for (size_t j = 0; j < TILE; ++j)
      c_t[i][j] = 0;

  __shared__ float a_s[S][L], b_s[S][L]; // khối A(L,S), khối B(S,L)

  // dịch chuyển khối A(L,S) tới hết hàng, và khối B(S,L) tới hết cột
  // A có kích cỡ M x N, B có kích cỡ N x P nên dùng chung biến k được
  for (size_t k = 0; k < N; k += S) {
    __syncthreads();
    // sA[:, :] = A[k : k + S, yblock : yblock + L];
    // sB[:, :] = B[k : k + S, xblock : xblock + L];
    for (size_t s = 0; s < S; ++s) {
      for (size_t l = 0; l < L; ++l) {
        a_s[s][l] = a[(yblock + l)*N +      (k + s)]; // a: M*N
        b_s[s][l] = a[     (k + s)*P + (xblock + l)]; // b: N*P
      }
    }
    __syncthreads();

    for (int ki = 0; ki < S; ++ki) {
      // Khởi tạo mảng a_t, b_t từ cột trong a_shared, và hàng b_shared
      for (size_t t = 0; t < TILE; ++t) { 
        // a[:] = sA[ki, threadIdx.y * V : threadIdx.y * V + V];
        // b[:] = sA[ki, threadIdx.x * V : threadIdx.x * V + V];
        a_t[t] = a_s[ki][threadIdx.y*TILE + t];
        b_t[t] = b_s[ki][threadIdx.x*TILE + t];
      }
      // Tính toán trên local vars
      for (size_t i = 0; i < TILE; ++i)
        for (size_t j = 0; j < TILE; ++j)
          c_t[i][j] += a_t[i] * b_t[j];
    }
  }

  const size_t ybase = yblock + threadIdx.y*TILE;
  const size_t xbase = xblock + threadIdx.y*TILE;
  // Update kết quả cho TILE * TILE tại ybase, xbase
  for (size_t i = 0; i < TILE; ++i)
    for (size_t j = 0; j < TILE; ++j)
      out[(ybase + i)*P + xbase + j] = c_t[i][j];
}

void Matmul(const CudaArray& a, const CudaArray& b, CudaArray* out,
  uint32_t M, uint32_t N, uint32_t P) {
  /**
   * Multiply two (compact) matrices into an output (also compact) matrix.
   * You will want to look at the lecture and notes on GPU-based linear algebra
   * to see how to do this. We would encourage you to use cooperative fetching, 
   * shared memory register tiling, and other ideas covered in the class notes.
   * 
   * Note that unlike the tiled matmul function in the CPU backend, here you should 
   * implement a single function that works across all size matrices, 
   * whether or not they are a multiple of a tile size. As with previous CUDA
   * implementations, this function here will largely just set up the kernel call,
   *  and you should implement the logic in a separate MatmulKernel() call.
   *
   * Args:
   *   a: compact 2D array of size m x n
   *   b: compact 2D array of size n x p
   *   out: compact 2D array of size m x p to write the output to
   *   M: rows of a / out
   *   N: columns of a / rows of b
   *   P: columns of b / out
   */

  /// BEGIN YOUR SOLUTION
  // if (false) { /*
  if (M % L == 0 && P % L == 0 && N % S == 0) {
    // => Can do shared-mem tiling
    // Mỗi thread tính (TILE, TILE) sub-matrix
    dim3 block(L / TILE, L / TILE, 1);
    dim3 grid(P / L, M / L, 1); // => M = blockDim.y * L, P = blockDim.x * L
    // (M/L)*(P/L)*(L/TILE)*(L/TILE) = (M*L)/(TILE*TILE) = out->size/(TILE*TILE)
    MatmulSharedMemKernel<<<grid, block>>>(a.ptr, b.ptr, out->ptr, P, N);
  /**/
  } else if (M % TILE == 0 && P % TILE == 0) {
    // Trường hợp M, P chia hết cho TILE thì dùng tile matmul
    size_t size = out->size / (TILE * TILE);
    CudaDims dim = CudaOneDim(size);
    MatmulTiledKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, 
        size, N, P);

  } else {
    // Nếu không thì dùng simple matmul mỗi thread tính 1 phần tử out[i,j]
    CudaDims dim = CudaOneDim(out->size);
    SimpleMatmulKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, M, N, P);
  }
  /// END YOUR SOLUTION
}

////////////////////////////////////////////////////////////////////////////////
// Max and sum reductions
////////////////////////////////////////////////////////////////////////////////

__global__ void ReduceMaxKernel(const scalar_t* a, scalar_t* out, size_t reduce_size, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) {
    /// BEGIN YOUR SOLUTION
    size_t offset = gid*reduce_size;
    scalar_t max = a[offset];
    for (size_t k = 1; k < reduce_size; k++) {
      const scalar_t tmp = a[offset + k];
      if (max < tmp) { max = tmp; }
    }
    out[gid] = max;
    /// END YOUR SOLUTION
  }
}

void ReduceMax(const CudaArray& a, CudaArray* out, size_t reduce_size) {
  /**
   * Reduce by taking maximum over `reduce_size` contiguous blocks.
   * Even though it is inefficient, for simplicity you can perform each reduction
   * in a single CUDA thread.
   * 
   * Args:
   *   a: compact array of size a.size = out.size * reduce_size to reduce over
   *   out: compact array to write into
   *   redice_size: size of the dimension to reduce over
   */
  /// BEGIN YOUR SOLUTION
  CudaDims dim = CudaOneDim(out->size);
  ReduceMaxKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, reduce_size, out->size);
  /// END YOUR SOLUTION
}



__global__ void ReduceSumKernel(const scalar_t* a, scalar_t* out, size_t reduce_size, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) {
    /// BEGIN YOUR SOLUTION
    size_t offset = gid*reduce_size;
    scalar_t sum = a[offset];
    for (size_t k = 1; k < reduce_size; k++) {
      sum += a[offset + k];
    }
    out[gid] = sum;
    /// END YOUR SOLUTION
  }
}

void ReduceSum(const CudaArray& a, CudaArray* out, size_t reduce_size) {
  /**
   * Reduce by taking summation over `reduce_size` contiguous blocks.
   * Again, for simplicity you can perform each reduction in a single CUDA thread.
   * 
   * Args:
   *   a: compact array of size a.size = out.size * reduce_size to reduce over
   *   out: compact array to write into
   *   redice_size: size of the dimension to reduce over
   */
  /// BEGIN YOUR SOLUTION
  CudaDims dim = CudaOneDim(out->size);
  ReduceSumKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, reduce_size, out->size);
  /// END YOUR SOLUTION
}

}  // namespace cuda
}  // namespace needle

PYBIND11_MODULE(ndarray_backend_cuda, m) {
  namespace py = pybind11;
  using namespace needle;
  using namespace cuda;

  m.attr("__device_name__") = "cuda";
  m.attr("__tile_size__") = TILE;

  py::class_<CudaArray>(m, "Array")
      .def(py::init<size_t>(), py::return_value_policy::take_ownership)
      .def_readonly("size", &CudaArray::size)
      .def("ptr", &CudaArray::ptr_as_int);

  // return numpy array, copying from GPU
  m.def("to_numpy", [](const CudaArray& a, std::vector<size_t> shape, 
    std::vector<size_t> strides, size_t offset) {
    // 
    std::vector<size_t> numpy_strides = strides;
    // biến đổi elems từ begin() tới end() và ghi vào bắt đầu từ begin()
    std::transform(numpy_strides.begin(), numpy_strides.end(), 
      numpy_strides.begin(), [](size_t& c) { return c * ELEM_SIZE; });

    /* copy memory to host */

    // khởi tạo vùng nhớ mới trong hosst
    scalar_t* host_ptr = (scalar_t*)std::malloc(a.size * ELEM_SIZE);
    if (host_ptr == 0) { throw std::bad_alloc(); }

    // a là CudaArray và a.ptr trỏ tới vùng nhớ trong GPU
    hipError_t err = hipMemcpy(
      host_ptr, a.ptr, a.size * ELEM_SIZE, hipMemcpyDeviceToHost);
    if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err));

    // return numpy array
    py::capsule deallocate_buffer(host_ptr, [](void* p) { free(p); });
    return py::array_t<scalar_t>(
      shape, numpy_strides, host_ptr + offset, deallocate_buffer);
  });


  // copy numpy array to GPU
  m.def("from_numpy", [](py::array_t<scalar_t> a, CudaArray* out) {
    hipError_t err = hipMemcpy(
      out->ptr, a.request().ptr, out->size * ELEM_SIZE, hipMemcpyHostToDevice);
    if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err));
  });


  m.def("fill", Fill);
  m.def("compact", Compact);
  m.def("ewise_setitem", EwiseSetitem);
  m.def("scalar_setitem", ScalarSetitem);
  m.def("ewise_add", EwiseAdd);
  m.def("scalar_add", ScalarAdd);

  m.def("ewise_mul", EwiseMul);
  m.def("scalar_mul", ScalarMul);
  m.def("ewise_div", EwiseDiv);
  m.def("scalar_div", ScalarDiv);
  m.def("scalar_power", ScalarPower);

  m.def("ewise_maximum", EwiseMaximum);
  m.def("scalar_maximum", ScalarMaximum);
  m.def("ewise_eq", EwiseEq);
  m.def("scalar_eq", ScalarEq);
  m.def("ewise_ge", EwiseGe);
  m.def("scalar_ge", ScalarGe);

  m.def("ewise_log", EwiseLog);
  m.def("ewise_exp", EwiseExp);
  m.def("ewise_tanh", EwiseTanh);

  m.def("matmul", Matmul);

  m.def("reduce_max", ReduceMax);
  m.def("reduce_sum", ReduceSum);
}